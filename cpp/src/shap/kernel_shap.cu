#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/explainer/kernel_shap.hpp>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace ML {
namespace Explainer {

/*
* Kernel distrubutes exact part of the kernel shap dataset
* Each block scatters the data of a row of `observations` into the (number of rows of
* background) in `dataset`, based on the row of `X`.
* So, given:
* background = [[0, 1, 2],
                [3, 4, 5]]
* observation = [100, 101, 102]
* X = [[1, 0, 1],
*      [0, 1, 1]]
*
* dataset (output):
* [[100, 1, 102],
*  [100, 4, 102]
*  [0, 101, 102],
*  [3, 101, 102]]
*
*
*/
template <typename DataT, typename IdxT>
__global__ void exact_rows_kernel_sm(DataT* X, IdxT nrows_X, IdxT ncols,
                                     DataT* background, IdxT nrows_background,
                                     DataT* dataset, DataT* observation) {
  extern __shared__ int idx[];
  int i, j;

  if (threadIdx.x < nrows_background) {
    // the first thread of each block gets the row of X that the block will use
    // for the scatter.
    if (threadIdx.x == 0) {
      for (i = 0; i < ncols; i++) {
        idx[i] = (int)X[blockIdx.x * ncols + i];
      }
    }
    __syncthreads();

    // all the threads now scatter the row, based on background and new observation
    int row = blockIdx.x * nrows_background + threadIdx.x;
#pragma unroll
    for (i = row; i < row + nrows_background; i += blockDim.x) {
#pragma unroll
      for (j = 0; j < ncols; j++) {
        if (idx[j] == 0) {
          dataset[i * ncols + j] = background[(i % nrows_background) * ncols + j];
        } else {
          dataset[i * ncols + j] = observation[j];
        }
      }
    }
  }
}

/*
* Similar kernel as above, but uses no shared memory for the index, in case
* it cannot fir in the shared memory of the device.
*
*/

template <typename DataT, typename IdxT>
__global__ void exact_rows_kernel(DataT* X, IdxT nrows_X, IdxT ncols,
                                  DataT* background, IdxT nrows_background,
                                  DataT* dataset, DataT* observation) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int i, j;

#pragma unroll
  for (i = tid; i < nrows_background; i += blockDim.x) {
#pragma unroll
    for (j = 0; j < ncols; j++) {
      if (X[blockIdx.x + j] == 0) {
        dataset[i * ncols + j] = background[(i % nrows_background) * ncols + j];
      } else {
        dataset[i * ncols + j] = observation[j];
      }
    }
  }
}

/*
* Kernel distrubutes sampled part of the kernel shap dataset
* The first thread of each block calculates the sampling of `k` entries of `observation`
* to scatter into `dataset`. Afterwards each block scatters the data of a row of `X` into the (number of rows of
* background) in `dataset`.
* So, given:
* background = [[0, 1, 2, 3],
                [5, 6, 7, 8]]
* observation = [100, 101, 102, 103]
* nsamples = [3, 2]
*
* X (output)
*      [[1, 0, 1, 1],
*       [0, 1, 1, 0]]
*
* dataset (output):
* [[100, 1, 102, 103],
*  [100, 6, 102, 103]
*  [0, 101, 102, 3],
*  [5, 101, 102, 8]]
*
*
*/
template <typename DataT, typename IdxT>
__global__ void sampled_rows_kernel(IdxT* nsamples, DataT* X, IdxT nrows_X,
                                    IdxT ncols, DataT* background,
                                    IdxT nrows_background, DataT* dataset,
                                    DataT* observation, uint64_t seed) {
  extern __shared__ int smps[];
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int i, j, k_blk;

  // see what k this block will generate
  k_blk = nsamples[blockIdx.x];

  if (threadIdx.x < nrows_background) {
    if (threadIdx.x == 0) {
      // thread 0 of block generates samples, reducing number of rng calls
      // calling hiprand only 3 * k times.
      // Sampling algo from: Li, Kim-Hung. "Reservoir-sampling algorithms
      // of time complexity O (n (1+ log (N/n)))." ACM Transactions on Mathematical
      // Software (TOMS) 20.4 (1994): 481-493.
      float w;
      hiprandState_t state;
      for (i = 0; i < k_blk; i++) {
        smps[i] = i;
      }
      hiprand_init((unsigned long long)seed, (unsigned long long)tid, 0, &state);

      w = exp(log(hiprand_uniform(&state)) / k_blk);

      while (i < ncols) {
        i = i + floor(log(hiprand_uniform(&state)) / log(1 - w)) + 1;
        if (i <= ncols) {
          smps[(int)(hiprand_uniform(&state) * k_blk)] = i;
          w = w * exp(log(hiprand_uniform(&state)) / k_blk);
        }
      }

      // write samples to 1-0 matrix
      for (i = 0; i < k_blk; i++) {
        X[i] = smps[i];
      }
    }

    // all threads write background line to their line

#pragma unroll
    for (i = tid; i < nrows_background; i += blockDim.x) {
#pragma unroll
      for (j = 0; j < ncols; j++) {
        dataset[i * ncols + j] = background[(i % nrows_background) * ncols + j];
      }
    }

    __syncthreads();

    // all threads write observation[samples] into their entry
#pragma unroll
    for (i = tid; i < nrows_background; i += blockDim.x) {
#pragma unroll
      for (j = 0; j < k_blk; j++) {
        dataset[i * ncols + smps[i]] = observation[smps[j]];
      }
    }
  }
}

template <typename DataT, typename IdxT>
void kernel_dataset_impl(const raft::handle_t& handle, DataT* X, IdxT nrows_X,
                         IdxT ncols, DataT* background, IdxT nrows_background,
                         DataT* combinations, DataT* observation, int* nsamples,
                         int len_samples, int maxsample, uint64_t seed) {
  const auto& handle_impl = handle;
  hipStream_t stream = handle_impl.get_stream();

  IdxT nblks;
  IdxT nthreads;

  // calculate how many threads per block we need in multiples of 32
  nthreads = std::min(int(nrows_background / 32 + 1) * 32, 512);

  // number of blocks for exact part of the dataset
  nblks = nrows_X - len_samples;

  hipDeviceProp_t prop;
  prop = handle_impl.get_device_properties();

  if (ncols * sizeof(DataT) <= prop.sharedMemPerMultiprocessor) {
    // each block calculates the combinations of an entry in X
    // at least nrows_background threads per block, multiple of 32
    exact_rows_kernel_sm<<<nblks, nthreads, ncols * sizeof(DataT), stream>>>(
      X, nrows_X, ncols, background, nrows_background, combinations,
      observation);
  } else {
    exact_rows_kernel<<<nblks, nthreads, 0, stream>>>(
      X, nrows_X, ncols, background, nrows_background, combinations,
      observation);
  }

  CUDA_CHECK(hipPeekAtLastError());

  // check if random part of the dataset  is needed
  if (len_samples > 0) {
    // each block does a sample
    nblks = len_samples;

    // shared memory shouldn't be a problem since k will be small
    // due to distribution of shapley kernel weights
    sampled_rows_kernel<<<nblks, nthreads, maxsample * sizeof(int), stream>>>(
      nsamples, &X[(nrows_X - len_samples) * ncols], len_samples, ncols,
      background, nrows_background, combinations, observation, seed);
  }

  CUDA_CHECK(hipPeekAtLastError());
}

void kernel_dataset(const raft::handle_t& handle, float* X, int nrows_X,
                    int ncols, float* background, int nrows_background,
                    float* dataset, float* observation, int* nsamples,
                    int len_nsamples, int maxsample, uint64_t seed) {
  kernel_dataset_impl(handle, X, nrows_X, ncols, background, nrows_background,
                      dataset, observation, nsamples, len_nsamples, maxsample,
                      seed);
}

void kernel_dataset(const raft::handle_t& handle, double* X, int nrows_X,
                    int ncols, double* background, int nrows_background,
                    double* dataset, double* observation, int* nsamples,
                    int len_nsamples, int maxsample, uint64_t seed) {
  kernel_dataset_impl(handle, X, nrows_X, ncols, background, nrows_background,
                      dataset, observation, nsamples, len_nsamples, maxsample,
                      seed);
}

}  // namespace Explainer
}  // namespace ML
